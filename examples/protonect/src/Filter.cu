#include "hip/hip_runtime.h"
/*
 * Filter.cu - a GPU program to perform threshold filtering
 */

#include <stdio.h>
#include <hip/hip_runtime.h>

#include "Filter.h"

#define BLOCK_SIZE 512

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors(err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
    if(hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
        exit(-1);
    }
}

__global__ 
void FilterKernel(const uchar *in, uchar* out, const int width, const uchar threshold)
{
	int idx =  blockIdx.y * width + blockIdx.x * BLOCK_SIZE + threadIdx.x;
	out[idx] = (in[idx] < threshold) ? in[idx] : 0;
}

void FilterGPU( const uchar* h_in, uchar* h_out, int height, int width, uchar threshold)
{
	int SIZE = height * width;
	hipEvent_t start, stop;

	checkCudaErrors( hipEventCreate(&start) );
	checkCudaErrors( hipEventCreate(&stop) );

	// Allocate the device input image
	uchar *d_in = NULL;
	checkCudaErrors( hipMalloc((void **)&d_in, SIZE) );

	// Allocate the device output image
	uchar *d_out = NULL;
	checkCudaErrors( hipMalloc((void **)&d_out, SIZE ) );

	// Copy the host input image  to the device memory
	checkCudaErrors( hipMemcpy(d_in, h_in, SIZE, hipMemcpyHostToDevice) );
	
	checkCudaErrors( hipEventRecord(start, NULL) );

	// Launch the CUDA Kernel
	dim3 block(BLOCK_SIZE);
	dim3 grid(width/BLOCK_SIZE, height);
	FilterKernel<<<grid, block>>>( d_in, d_out, width, threshold);
	checkCudaErrors( hipGetLastError() );

	checkCudaErrors( hipEventRecord(stop, NULL) );
	checkCudaErrors( hipEventSynchronize(stop) );

	// Copy the device result to the host
	checkCudaErrors( hipMemcpy(h_out, d_out, SIZE, hipMemcpyDeviceToHost) );

	checkCudaErrors( hipFree(d_in) );
	checkCudaErrors( hipFree(d_out) );

	float msec = 0.f;
	checkCudaErrors( hipEventElapsedTime(&msec, start, stop) );

	printf("GPU code ran for: %f ms\n", msec);
}
