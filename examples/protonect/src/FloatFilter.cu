#include "hip/hip_runtime.h"
/*
 * Filter.cu - a GPU program to perform threshold filtering
 */

#include <stdio.h>
#include <hip/hip_runtime.h>

#include "Filter.h"

#define BLOCK_SIZE 512

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error
#define checkCudaErrors(err)  __checkCudaErrors(err, __FILE__, __LINE__)

inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
{
    if(hipSuccess != err)
    {
        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
        exit(-1);
    }
}

__global__ 
void FilterKernel(const float* in, float* out, const int width, const float threshold)
{
	int idx =  blockIdx.y * width + blockIdx.x * BLOCK_SIZE + threadIdx.x;
        out[idx] = (in[idx] < threshold) ? in[idx] : 0;
}

void FilterGPU( const float* h_in, float* h_out, int height, int width, float threshold, bool verbose)
{
	int SIZE = height * width * sizeof(float);
	hipEvent_t start, stop;

	checkCudaErrors( hipEventCreate(&start) );
	checkCudaErrors( hipEventCreate(&stop) );

	// Allocate the device input image
	float* d_in = NULL;
	checkCudaErrors( hipMalloc((void **)&d_in, SIZE) );

	// Allocate the device output image
	float *d_out = NULL;
	checkCudaErrors( hipMalloc((void **)&d_out, SIZE) );

	// Copy the host input image  to the device memory
	checkCudaErrors( hipMemcpy(d_in, h_in, SIZE, hipMemcpyHostToDevice) );
	
	checkCudaErrors( hipEventRecord(start, NULL) );

	// Launch the CUDA Kernel
	dim3 block(BLOCK_SIZE);
	dim3 grid(width/BLOCK_SIZE, height);
	FilterKernel<<<grid, block>>>( d_in, d_out, width, threshold);
	checkCudaErrors( hipGetLastError() );

	checkCudaErrors( hipEventRecord(stop, NULL) );
	checkCudaErrors( hipEventSynchronize(stop) );

	// Copy the device result to the host
	checkCudaErrors( hipMemcpy(h_out, d_out, SIZE, hipMemcpyDeviceToHost) );

	checkCudaErrors( hipFree(d_in) );
	checkCudaErrors( hipFree(d_out) );

	float msec = 0.f;
	checkCudaErrors( hipEventElapsedTime(&msec, start, stop) );

	if (verbose)
		printf("GPU code ran for: %f ms\n", msec);
}
